#include "hip/hip_runtime.h"
#include "blake2.h"
#include "bc_miner.h"

#include <string>
#include <vector>
#include <algorithm>
#include <ctime>
#include <pthread.h>
#include <assert.h>
#include <chrono>
#include <random>

#include <iostream>

/*
INFO	 mining.thread worker 4517 reporting in 
INFO	 mining.primitives twork: 7296c034e95304ee2a69c2a61e6287a0 58448d08d723222d19658c0364dd247c 64
                                  58448d08d723222d19658c0364dd247c
INFO	 mining.primitives miner: 0xf34fa87db39d15471bebe997860dcd49fc259318 42 
INFO	 mining.primitives merkl: 108459d41ce2399e13992528ea1bd9940fac9df181e29c2b61caeaad55f6532a 64 
INFO	 mining.primitives nhash: 1af153f4cf971b61cc867b760bebfbc98a8c216cd69d6bb5dbb63aaed9db1fc3 64 
INFO	 mining.primitives times: 1536810114 10 
INFO	 mining.primitives cocat: 0xf34fa87db39d15471bebe997860dcd49fc259318108459d41ce2399e13992528ea1bd9940fac9df181e29c2b61caeaad55f6532a1af153f4cf971b61cc867b760bebfbc98a8c216cd69d6bb5dbb63aaed9db1fc31536810114 180 
INFO	 mining.primitives solun: 2b08d9146a6ce1f02db88203bdc653cd0b5e33ec945f391f82dbe3d417fb586e 64 
INFO	 mining.primitives wrkck: 53,56,52,52,56,100,48,56,100,55,50,51,50,50,50,100,49,57,54,53,56,99,48,51,54,52,100,100,50,52,55,99, 55,50,57,54,99,48,51,52,101,57,53,51,48,52,101,101,50,97,54,57,99,50,97,54,49,101,54,50,56,55,97,48 2 
INFO	 mining.primitives compr: 2b08d9146a6ce1f02db88203bdc653cd0b5e33ec945f391f82dbe3d417fb586e 64 
INFO	 mining.primitives testr: 204933315567342 undefined
*/

uint64_t mypow(uint64_t base, uint64_t exp) {
  int result = 1;
  while( exp-- ) { result *= base; }
  return result;
}

struct sort_by_distance {
  const size_t* distances;
  bool operator()(size_t i1,size_t i2) const {
    //std::cout << i1 << ' ' << distances[i1] << " >?= " << i2 << ' ' << distances[i2] << std::endl;
    return distances[i1] >= distances[i2]; 
  }
};

int main(int argc, char **argv) {
    
    std::string work ("7ca44f0c6f416240157a7d9067802269a64ab5503bd11970e3130d36e75ab815");
    std::string mhash("0xc95d2b8fe219f528d10cc35d0df78da90a32a8a9");
              //mhash("0xbfcf55b8fcb3d1937a1c6d02ff6d17089651882c");
	      //mhash("0xf34fa87db39d15471bebe997860dcd49fc259318");
    std::string merkl("d1dde6972fa183f1b9ca2c0ee2d6d87656be16db28c7d6cc3c5086eb7fda8fe5");
    uint64_t thenonce = std::numeric_limits<uint64_t>::max();
    uint8_t nonce_string[22]; // ten digits and a null character max;
    memset(nonce_string,0,22);
    // convert nonce
    static uint16_t num_to_code[16] = {48,49,50,51,52,53,54,55,56,57};
    nonce_string[0] = '0'; // take care of base case
    int length = 0;
    uint64_t red_nonce = thenonce;
    while( red_nonce > 0 ) { ++length; red_nonce /= 10ULL; }
    red_nonce = thenonce;
    std::cout << "the length: " << length << std::endl;
    for( uint64_t i = length; i > 1; --i ) {
      nonce_string[i-1] = num_to_code[red_nonce%10];
      red_nonce /= 10ULL;
    }
    nonce_string[0] = num_to_code[red_nonce];
    std::cout << thenonce << ' ' << nonce_string << ' ' << red_nonce << std::endl;    
    std::string nhash("cb5d17fe5c27f7b7426002eb665142d00190553b9d945a936eed3ffd23cdde71");
    std::string times("1585772256");

    std::string the_thing = mhash + merkl + nhash + times;

    std::string result_bc("c0d42acc9793a81096411b74b78fe9a12645737c57ee1544fb35d5fa6f09503e");
        
    // now let's do it on the GPU for real
    size_t stash_size = mhash.length();
    size_t tstamp_size = times.length();
        
    std::vector<bc_mining_stream> streams;    

    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();

    std::mt19937 generator(seed);

    init_gpus(streams);

    std::vector<pthread_t> threads(streams.size());
    std::vector<bc_thread_data> thread_data(streams.size());

    bc_mining_inputs* in; 
    hipHostMalloc(&in,streams.size()*sizeof(bc_mining_inputs));
    bc_mining_outputs* out;
    hipHostMalloc(&out,streams.size()*sizeof(bc_mining_outputs));

    uint64_t start_nonce = generator();
    bool solution_found = false;
    for(unsigned iGPU = 0; iGPU < streams.size(); ++iGPU ) {
      in[iGPU].miner_key_size_ = mhash.length();
      in[iGPU].time_stamp_size_ = times.length();
      in[iGPU].work_size_ = work.length();    
      in[iGPU].the_difficulty_ = 315874869807223ULL;
      
      memcpy(in[iGPU].miner_key_,mhash.c_str(),in[iGPU].miner_key_size_);
      memcpy(in[iGPU].merkel_root_,merkl.c_str(),BLAKE2B_OUTBYTES);
      memcpy(in[iGPU].time_stamp_,times.c_str(),in[iGPU].time_stamp_size_);
      //set the work
      for(unsigned i = 0; i < in[iGPU].work_size_; ++i ) {
	char temp[2];
	temp[0] = work[i];
	temp[1] = '\0';
	in[iGPU].received_work_[i/2] += strtol(temp,NULL,16)<<(4*((i+1)%2));
      }

      thread_data[iGPU].in = in + iGPU;
      thread_data[iGPU].out = out + iGPU;
      thread_data[iGPU].stream = &streams[iGPU];
      thread_data[iGPU].start_nonce = std::numeric_limits<uint64_t>::max() - iGPU*100*HASH_TRIES; //start_nonce + 100*HASH_TRIES*iGPU + generator();
      thread_data[iGPU].solution_found = &solution_found;
      std::cout<< iGPU << ' ' << thread_data[iGPU].start_nonce << std::endl;
    }

    int result_code;
    for( unsigned iGPU = 0; iGPU < streams.size(); ++iGPU ) {      
      result_code = pthread_create(&threads[iGPU], NULL, run_miner_thread, &thread_data[iGPU]);
      assert(!result_code);
    }
    
    
    for ( unsigned iGPU = 0; iGPU < streams.size(); ++iGPU) {
      // block until thread 'index' completes
      result_code = pthread_join(threads[iGPU], NULL);
      assert(!result_code);
      std::cout << "In main: thread " << iGPU <<" has completed" << std::endl;
    }

    for( unsigned iGPU = 0; iGPU < streams.size(); ++iGPU ) {
      std::cout << "gpu: " << streams[iGPU].device << " trial = 0x" << std::hex;
      // output "blake2bl"
      for( unsigned i = 32; i < BLAKE2B_OUTBYTES; ++i ) {
	std::cout << std::hex << (unsigned)(out[iGPU].result_blake2b_[i]>>4) << (unsigned)(out[iGPU].result_blake2b_[i]&0xf);
      }
      std::cout << std::dec << std::endl;
      std::cout << "gpu distance is: " << out[iGPU].distance_ << std::endl;
      std::cout << "gpu nonce is   : " << out[iGPU].nonce_ << std::endl;
      std::cout << "gpu iterations : " << out[iGPU].iterations_ << std::endl;
    }

    destroy_gpus(streams);
    
    hipHostFree(in);
    hipHostFree(out);
    
    return 0;
}
